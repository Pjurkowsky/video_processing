#include "hip/hip_runtime.h"
#include <cuda.cuh>

__global__ void add(float* a, float* b) {
  *a = *a + *b;
}

float add_gpu(float a, float b) {
  float *x, *y;
  float res = 0.0;
  hipMallocManaged(&x, sizeof(float));
  hipMallocManaged(&y, sizeof(float));
  *x = a;
  *y = b;
  add<<<1, 1>>>(x, y);
  hipDeviceSynchronize();
  res = *x;
  hipFree(x);
  hipFree(y);
  return res;
}
