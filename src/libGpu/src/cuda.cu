#include "hip/hip_runtime.h"
#include <common.h>
#include <cstdint>
#include <cstring>
#include <cuda.cuh>
#include <cuda.hpp>

__global__ void cuda::bgr_to_mono(uint8_t* frame, int height, int width) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int row = index; row < height; row += stride) {
    for (int col = 0; col < width; col++) {
      int index = 3 * (row * width + col);
      uint8_t avg = (frame[index] + frame[index + 1] + frame[index + 2]) / 3;
      frame[index] = avg;
      frame[index + 1] = avg;
      frame[index + 2] = avg;
    }
  }
}

void gpu::bgr_to_mono(uint8_t* frame, int height, int width) {
  uint8_t* buffer;
  int buffer_size = sizeof(uint8_t) * width * height * 3;

  hipMalloc(&buffer, buffer_size);
  hipMemcpy(buffer, frame, buffer_size, hipMemcpyKind::hipMemcpyHostToDevice);
  cuda::bgr_to_mono<<<16, 256>>>(buffer, height, width);
  hipDeviceSynchronize();
  hipMemcpy(frame, buffer, buffer_size, hipMemcpyKind::hipMemcpyDeviceToHost);
  hipFree(buffer);
}

void gpu::bgr_to_mono(uint8_t* frames, int frame_count, int height, int width) {

  uint8_t* buffer;
  int frame_size = sizeof(uint8_t) * width * height * 3;
  hipMalloc(&buffer, frame_size);

  for (int i = 0; i < 1; i++) {
    hipMemcpy(buffer, frames + frame_size * i, frame_size, hipMemcpyKind::hipMemcpyHostToDevice);
    cuda::bgr_to_mono<<<16, 256>>>(buffer, height, width);
    hipDeviceSynchronize();
    hipMemcpy(frames + frame_size * i, buffer, frame_size, hipMemcpyKind::hipMemcpyDeviceToHost);
  }

  hipFree(buffer);
}

void gpu::init() {
  hipFree(0);
}
