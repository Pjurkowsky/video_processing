#include "hip/hip_runtime.h"
#include <common.h>
#include <cstdint>
#include <cstring>
#include <cuda.cuh>
#include <cuda.hpp>
#include <logger.h>

#define LOG(level) TaggedLogStream("CUDA", level)

#define ERROR_LOG LOG(LEVEL_ERROR)
#define INFO_LOG LOG(LEVEL_INFO)
#define WARNING_LOG LOG(LEVEL_WARNING)

__global__ void cuda::bgr_to_mono(uint8_t *frame, int height, int width) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < height && col < width) {
    int index = 3 * (row * width + col);
    uint8_t avg = (frame[index] + frame[index + 1] + frame[index + 2]) / 3;
    frame[index] = avg;
    frame[index + 1] = avg;
    frame[index + 2] = avg;
  }
}

void gpu::bgr_to_mono(uint8_t *frame, int height, int width) {
  uint8_t *buffer;
  int buffer_size = sizeof(uint8_t) * width * height * 3;

  hipMalloc(&buffer, buffer_size);
  hipMemcpy(buffer, frame, buffer_size,
             hipMemcpyKind::hipMemcpyHostToDevice);
  cuda::bgr_to_mono<<<16, 256>>>(buffer, height, width);
  hipDeviceSynchronize();
  hipMemcpy(frame, buffer, buffer_size,
             hipMemcpyKind::hipMemcpyDeviceToHost);
  hipFree(buffer);
}

void gpu::bgr_to_mono(uint8_t *frames, int batch_size, int height, int width,
                      uint8_t *buffer) {
  int frame_size = sizeof(uint8_t) * width * height * 3;
  int total_size = frame_size * batch_size;

  hipMemcpy(buffer, frames, total_size,
             hipMemcpyKind::hipMemcpyHostToDevice);
  for (int i = 0; i < batch_size; i++) {
    uint8_t *frame_ptr = buffer + i * frame_size;
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    cuda::bgr_to_mono<<<grid, block>>>(frame_ptr, height, width);
  }
  hipDeviceSynchronize();
  hipMemcpy(frames, buffer, total_size,
             hipMemcpyKind::hipMemcpyDeviceToHost);
}

void gpu::malloc_memory(uint8_t **src_buffer, int buffer_size) {
  hipMalloc(src_buffer, buffer_size);
}

void gpu::free_memory(uint8_t *buffer) { hipFree(buffer); }

void gpu::resize(uint8_t *frames, int batch_size, int src_height, int src_width,
                 int dst_height, int dst_width, uint8_t *resized_frames,
                 uint8_t *src_buffer, uint8_t *dst_buffer) {
  int frame_size = sizeof(uint8_t) * src_width * src_height * 3;
  int resized_frame_size = sizeof(uint8_t) * dst_width * dst_height * 3;
  int total_size = frame_size * batch_size;

  // hipMemcpy(src_buffer, frames, total_size, hipMemcpyHostToDevice);

  for (int i = 0; i < batch_size; i++) {
    uint8_t *src_frame = src_buffer + i * frame_size;
    uint8_t *dst_frame = dst_buffer + i * resized_frame_size;
    dim3 block(16, 16);
    dim3 grid((dst_width + 15) / 16, (dst_height + 15) / 16);
    cuda::resize_kernel<<<grid, block>>>(src_frame, dst_frame, src_height,
                                         src_width, dst_height, dst_width);
  }

  hipDeviceSynchronize();

  // hipMemcpy(resized_frames, dst_buffer, resized_frame_size * batch_size,
  //           hipMemcpyDeviceToHost);
}

// nearest neighbour algorithm
__global__ void cuda::resize_kernel(uint8_t *src_frame, uint8_t *dst_frame,
                                    int src_height, int src_width,
                                    int dst_height, int dst_width) {
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;

  int channel = 3;

  if (row < dst_height && col < dst_width) {
    int rowIn = row * src_height / dst_height;
    int colIn = col * src_width / dst_width;

    dst_frame[(row * dst_width + col) * channel + 0] =
        src_frame[(rowIn * src_width + colIn) * channel + 0];
    dst_frame[(row * dst_width + col) * channel + 1] =
        src_frame[(rowIn * src_width + colIn) * channel + 1];
    dst_frame[(row * dst_width + col) * channel + 2] =
        src_frame[(rowIn * src_width + colIn) * channel + 2];
  }
}

void gpu::init() {
  INFO_LOG << "Initializing CUDA context";
  hipFree(0);
}
void gpu::copyDeviceDevice(uint8_t *source, uint8_t *target, int size) {
  hipMemcpy(target, source, size, hipMemcpyKind::hipMemcpyDeviceToDevice);
}
