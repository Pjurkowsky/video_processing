#include "hip/hip_runtime.h"
#include <common.h>
#include <cstdint>
#include <cstring>
#include <cuda.cuh>
#include <cuda.hpp>

__global__ void cuda::bgr_to_mono(uint8_t* frame, int height, int width) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int row = index; row < height; row += stride) {
    for (int col = 0; col < width; col++) {
      int index = 3 * (row * width + col);
      uint8_t avg = (frame[index] + frame[index + 1] + frame[index + 2]) / 3;
      frame[index] = avg;
      frame[index + 1] = avg;
      frame[index + 2] = avg;
    }
  }
}

void gpu::bgr_to_mono(uint8_t* frame, int height, int width) {
  uint8_t* buffer;
  int buffer_size = sizeof(uint8_t) * width * height * 3;

  hipMalloc(&buffer, buffer_size);
  hipMemcpy(buffer, frame, buffer_size, hipMemcpyKind::hipMemcpyHostToDevice);
  cuda::bgr_to_mono<<<16, 256>>>(buffer, height, width);
  hipDeviceSynchronize();
  hipMemcpy(frame, buffer, buffer_size, hipMemcpyKind::hipMemcpyDeviceToHost);
  hipFree(buffer);
}

void gpu::bgr_to_mono(uint8_t* frames, int batch_size, int height, int width) {
  uint8_t* buffer;
  int frame_size = sizeof(uint8_t) * width * height * 3;
  int total_size = frame_size * batch_size;
  hipMalloc(&buffer, total_size);

    hipMemcpy(buffer, frames, total_size, hipMemcpyKind::hipMemcpyHostToDevice);
    for (int i = 0; i < batch_size; i++) {
      uint8_t* frame_ptr = buffer + i * frame_size;
      cuda::bgr_to_mono<<<16, 256>>>(frame_ptr, height, width);
    }
    hipDeviceSynchronize();
    hipMemcpy(frames, buffer, total_size, hipMemcpyKind::hipMemcpyDeviceToHost);
  
  hipFree(buffer);
}

void gpu::resize(uint8_t* frames, int batch_size, int src_height, int src_width, int dst_height, int dst_width, uint8_t* resized_frames) {
    uint8_t* src_buffer;
    uint8_t* dst_buffer;
    int frame_size = sizeof(uint8_t) * src_width * src_height * 3;
    int resized_frame_size = sizeof(uint8_t) * dst_width * dst_height * 3;
    int total_size = frame_size * batch_size;

    hipMalloc(&src_buffer, total_size);
    hipMalloc(&dst_buffer, resized_frame_size * batch_size);

    hipMemcpy(src_buffer, frames, total_size, hipMemcpyHostToDevice);

    for (int i = 0; i < batch_size; i++) {
        uint8_t* src_frame = src_buffer + i * frame_size;
        uint8_t* dst_frame = dst_buffer + i * resized_frame_size;
        dim3 block(16, 16);
        dim3 grid((dst_width + 15) / 16, (dst_height + 15) / 16);
        cuda::resize_kernel<<<grid, block>>>(src_frame, dst_frame, src_height, src_width, dst_height, dst_width);
    }

    hipDeviceSynchronize();

    hipMemcpy(resized_frames, dst_buffer, resized_frame_size * batch_size, hipMemcpyDeviceToHost);

    hipFree(src_buffer);
    hipFree(dst_buffer);
}

// nearest neighbour algorithm
__global__ void cuda::resize_kernel(uint8_t* src_frame, uint8_t* dst_frame, int src_height, int src_width, int dst_height, int dst_width) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    int channel = 3;

    if (i < dst_height && j < dst_width) {
        int iIn = i * src_height / dst_height;
        int jIn = j * src_width / dst_width;

        dst_frame[(i * dst_width + j) * channel + 0] = src_frame[(iIn * src_width + jIn) * channel + 0];
        dst_frame[(i * dst_width + j) * channel + 1] = src_frame[(iIn * src_width + jIn) * channel + 1];
        dst_frame[(i * dst_width + j) * channel + 2] = src_frame[(iIn * src_width + jIn) * channel + 2];
    }
}

void gpu::init() {
  hipFree(0);
}
