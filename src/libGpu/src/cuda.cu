#include "hip/hip_runtime.h"
#include <common.h>
#include <cstdint>
#include <cstring>
#include <cuda.cuh>
#include <cuda.hpp>

__global__ void cuda::bgr_to_mono(uint8_t* frame, int height, int width) {
  for (int row = 0; row < height; row++) {
    for (int col = 0; col < width; col++) {
      int index = 3 * (row * width + col);
      uint8_t avg = (frame[index] + frame[index + 1] + frame[index + 2]) / 3;
      frame[index] = avg;
      frame[index + 1] = avg;
      frame[index + 2] = avg;
    }
  }
}

void gpu::bgr_to_mono(uint8_t* frame, int height, int width) {
  uint8_t* buffer;
  int buffer_size = sizeof(uint8_t) * width * height * 3;

  hipMallocManaged(&buffer, buffer_size);
  memcpy(buffer, frame, buffer_size);
  benchmark([&buffer, &width, &height]() {
    cuda::bgr_to_mono<<<1, 1>>>(buffer, height, width);
  });
  hipDeviceSynchronize();

  memcpy(frame, buffer, buffer_size);
  hipFree(buffer);
}
